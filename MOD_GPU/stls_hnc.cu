#include "hip/hip_runtime.h"
extern "C" {
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include "solvers.h"
#include "stls.h"
#include "stls_hnc.h"
}

// -------------------------------------------------------------------
// FUNCTION USED TO ITERATIVELY SOLVE THE STLS-HNC EQUATIONS
// -------------------------------------------------------------------

void solve_stls_hnc(input in, bool verbose, bool iet) {

  // Arrays for STLS solution
  float *xx = NULL; 
  float *phi = NULL;
  float *GG = NULL;
  float *GG_new = NULL;
  float *SS = NULL;
  float *SSHF = NULL;
  float *bf = NULL;

  // Allocate arrays
  alloc_stls_arrays(in, &xx, &phi, &GG, &GG_new, &SS, &SSHF);

  // Initialize arrays that are not modified with the iterative procedure
  init_fixed_stls_arrays(&in, xx, phi, SSHF, verbose);
  bf = (float*)malloc( sizeof(float) * in.nx);
  compute_bf(bf, xx, in, iet);

  // Initial guess for Static structure factor (SSF) and static-local field correction (SLFC)
  if (strcmp(in.guess_file,"NO_FILE")==0){
    for (int ii=0; ii < in.nx; ii++) {
      GG[ii] = 0.0;
      GG_new[ii] = 1.0;
    }
    compute_ssf(SS, SSHF, GG, phi, xx, in);
  }
  else {
    read_guess(SS, GG, in);
  }
   
  // SSF and SLFC via iterative procedure
  if (verbose) printf("SSF and SLFC calculation...\n");
  float iter_err = 1.0;
  int iter_counter = 0;
  while (iter_counter < in.nIter && iter_err > in.err_min_iter ) {
    
    // Start timing
    clock_t tic = clock();
    
    // Update SSF
    compute_ssf(SS, SSHF, GG, phi, xx, in);
    
    // Update SLFC
    compute_slfc_hnc(GG_new, GG, SS, bf, xx, in);
    
    // Update diagnostic
    iter_err = 0.0;
    iter_counter++;
    for (int ii=0; ii<in.nx; ii++) {
      iter_err += (GG_new[ii] - GG[ii]) * (GG_new[ii] - GG[ii]);
      GG[ii] = in.a_mix*GG_new[ii] + (1-in.a_mix)*GG[ii];
    }
    iter_err = sqrt(iter_err);
    
    // End timing
    clock_t toc = clock();

    // Print diagnostic
    if (verbose) {
      printf("--- iteration %d ---\n", iter_counter);
      printf("Elapsed time: %f seconds\n", ((float)toc - (float)tic) / CLOCKS_PER_SEC);
      printf("Residual error: %.5e\n", iter_err);
      fflush(stdout);
    }
  }
  if (verbose) printf("Done.\n");
  
  // Internal energy
  if (verbose) printf("Internal energy: %f\n",compute_uex(SS, in));
  
  // Output to file
  if (verbose) printf("Writing output files...\n");
  write_text(SS, GG, phi, SSHF, xx, in);
  write_guess(SS, GG, in);
  if (verbose) printf("Done.\n");

  // Free memory
  free_stls_arrays(xx, phi, GG, GG_new, SS, SSHF);
  free(bf);
 
}


// -------------------------------------------------------------------
// FUNCTION USED TO COMPUTE THE STATIC LOCAL FIELD CORRECTION
// -------------------------------------------------------------------

void compute_slfc_hnc(float *GG_new, float *GG, float *SS,
		      float *bf, float *xx, input in) {

  // Static local field correction
  for (int ii=0; ii<in.nx; ii++) {

    float kmax, kmin, fu, 
    xx2, uu, uu2, ww2, ww;
   
    xx2 = xx[ii]*xx[ii];
    GG_new[ii] = 0.0;
    
    for (int jj=0; jj<in.nx; jj++){

      uu = xx[jj];
      uu2 = uu*uu;
      kmin = ii-jj;
      if (kmin < 0) kmin = -kmin;
      kmax = ii+jj;
      if (kmax > in.nx) kmax = in.nx;
      
      fu = 0.0;
      for (int kk=kmin; kk<kmax; kk++){

	ww = xx[kk];
	ww2 = ww*ww;
	fu += (ww2 - uu2 - xx2) * ww * (SS[kk] - 1.0);
	
      }
      
      GG_new[ii] += (1.0 - (GG[jj] - 1.0) * (SS[jj] - 1.0) 
		     - bf[jj])*in.dx*fu/uu;
      
    }
    
    GG_new[ii] *= 3.0*in.dx/(8.0*xx[ii]);
    GG_new[ii] += bf[ii];
    
  }

}

void compute_bf(float *bf, float *xx, input in, bool iet){

  float scaling = 1.0;
  float ll = pow(4.0/(9.0*M_PI), 1.0/3.0);
  float l2 = ll*ll, l3 = l2*ll, l4 = l3*ll, l5 = l4*ll, 
    l6 = l5*ll, l7 = l6*ll, l8 = l7*ll;
  float Gamma = scaling*2*l2*in.rs/in.Theta;
  float lnG = log(Gamma), lnG2 = lnG*lnG;
  float b0 = 0.258 - 0.0612*lnG + 0.0123*lnG2 - 1.0/Gamma;
  float b1 = 0.0269 + 0.0318*lnG + 0.00814*lnG2;
  float c1 = 0.498 - 0.280*lnG + 0.0294*lnG2;
  float c2 = -0.412 + 0.219*lnG - 0.0251*lnG2;
  float c3 = 0.0988 - 0.0534*lnG + 0.00682*lnG2;
  float b02 = b0*b0, b03 = b02*b0, b04 = b03*b0, b05 = b04*b0,
    b06 = b05*b0, b07 = b06*b0, b08 = b07*b0;
  float b12 = b1*b1, b13 = b12*b1, b14 = b13*b1, b15 = b14*b1,
    b16 = b15*b1, b17 = b16*b1, b18 = b17*b1;
  float b02_b12 = b02/b12, b03_b13 = b03/b13, b04_b14 = b04/b14,
    b05_b15 = b05/b15, b06_b16 = b06/b16, b07_b17 = b07/b17, 
    b08_b18 = b08/b18;
  float ff = 0.0;
  float q2,q3,q4,q5,q6,q7,q8;
  float bf1, bf2, bf3;

  if (iet){
    if (b0/b1 >= 0.0)
      ff = sqrt(M_PI)/(4.0*l2)*pow(b0/b1, 1.5);
    else{
      printf("Error: The STLS-IET scheme cannot be applied to this state point"
	     "(Gamma = %.8f) because the bridge function term diverges\n", Gamma);
      exit(EXIT_FAILURE);
    }
  }

  for (int ii=0; ii<in.nx; ii++){

    if (iet){
      q2 = xx[ii]*xx[ii];
      q3 = q2*xx[ii];
      q4 = q3*xx[ii];
      q5 = q4*xx[ii];
      q6 = q5*xx[ii];
      q7 = q6*xx[ii]; 
      q8 = q7*xx[ii];
      bf1 = -b0 + c1/16.0*(60.0*b02_b12 - 20.0*b03_b13*q2/l2 + b04_b14*q4/l4);
      bf2 = c2/64.0*(840.0*b03_b13 - 420.0*b04_b14*q2/l2 +
		     42.0*b05_b15*q4/l4 - b06_b16*q6/l6);
      bf3 = c3/256.0*(15120.0*b04_b14 - 10080.0*b05_b15*q2/l2 +
		     1512.0*b06_b16*q4/l4 - 72.0*b07_b17*q6/l6 + 
		     b08_b18*q8/l8);
      bf[ii] = scaling*ff*q2*(bf1 + bf2 + bf3)*exp(-b0*q2/(4.0*b1*l2));
    }
    else 
      bf[ii] = 0.0;
      
  }

}
