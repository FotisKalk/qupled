#include "hip/hip_runtime.h"
extern "C" {
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include "solvers.h"
#include "chemical_potential.h"
#include "stls.h"
}

// -------------------------------------------------------------------
// FUNCTION USED TO ITERATIVELY SOLVE THE STLS EQUATIONS
// -------------------------------------------------------------------

void solve_stls(input in, bool verbose) {

  // Arrays for STLS solution
  float *xx = NULL; 
  float *phi = NULL;
  float *GG = NULL;
  float *GG_new = NULL;
  float *SS = NULL;
  float *SSHF = NULL;

  // Allocate arrays
  alloc_stls_arrays(in, &xx, &phi, &GG, &GG_new, &SS, &SSHF);

  // Initialize arrays that are not modified with the iterative procedure
  init_fixed_stls_arrays(&in, xx, phi, SSHF, verbose);
  
  // Initial guess for Static structure factor (SSF) and static-local field correction (SLFC)
  if (strcmp(in.guess_file,"NO_FILE")==0){
    for (int ii=0; ii < in.nx; ii++) {
      GG[ii] = 0.0;
      GG_new[ii] = 1.0;
    }
    compute_ssf(SS, SSHF, GG, phi, xx, in);
  }
  else {
    read_guess(SS, GG, in);
  }


  // SSF and SLFC via iterative procedure
  if (verbose) printf("SSF and SLFC calculation...\n");
  float iter_err = 1.0;
  int iter_counter = 0;
  while (iter_counter < in.nIter && iter_err > in.err_min_iter ) {
    
    // Start timing
    clock_t tic = clock();
    
    // Update SSF
    compute_ssf(SS, SSHF, GG, phi, xx, in);

    // Update SLFC
    compute_slfc(GG_new, SS, xx, in);
    
    // Update diagnostic
    iter_err = 0.0;
    iter_counter++;
    for (int ii=0; ii<in.nx; ii++) {
      iter_err += (GG_new[ii] - GG[ii]) * (GG_new[ii] - GG[ii]);
      GG[ii] = in.a_mix*GG_new[ii] + (1-in.a_mix)*GG[ii];
    }
    iter_err = sqrt(iter_err);  
   
    // End timing
    clock_t toc = clock();

    // Print diagnostic
    if (verbose) {
      printf("--- iteration %d ---\n", iter_counter);
      printf("Elapsed time: %f seconds\n", ((float)toc - (float)tic) / CLOCKS_PER_SEC);
      printf("Residual error: %.5e\n", iter_err);
      fflush(stdout);
    }
  }
  if (verbose) printf("Done.\n");
  
  // Internal energy
  if (verbose) printf("Internal energy: %f\n",compute_uex(SS, in));
  
  // Output to file
  if (verbose) printf("Writing output files...\n");
  write_text(SS, GG, phi, SSHF, xx, in);
  write_guess(SS, GG, in); 
  if (verbose) printf("Done.\n");

  // Free memory
  free_stls_arrays(xx, phi, GG, GG_new, SS, SSHF);

 
}

// -------------------------------------------------------------------
// FUNCTIONS USED TO ALLOCATE AND FREE ARRAYS
// -------------------------------------------------------------------

void alloc_stls_arrays(input in, float **xx, float **phi, 
		       float **GG, float **GG_new, 
		       float **SS, float **SSHF){

  *xx = (float*)malloc( sizeof(float) * in.nx);
  *phi = (float*)malloc( sizeof(float) * in.nx * in.nl);
  *SSHF = (float*)malloc( sizeof(float) * in.nx);
  *GG = (float*)malloc( sizeof(float) * in.nx);
  *GG_new = (float*)malloc( sizeof(float) * in.nx);
  *SS = (float*)malloc( sizeof(float) * in.nx);
  
}

void free_stls_arrays(float *xx, float *phi, float *GG, 
		      float *GG_new, float *SS,
		      float *SSHF){

  free(xx);
  free(phi);
  free(SSHF);
  free(SS);
  free(GG);
  free(GG_new);
 
}


// -------------------------------------------------------------------
// FUNCTION USED TO INITIALIZE ARRAYS
// -------------------------------------------------------------------

void init_fixed_stls_arrays(input *in, float *xx, 
			    float *phi, float *SSHF, bool verbose){

  // Print on screen the parameter used to solve the STLS equation
  printf("------ Parameters used in the solution -------------\n");
  printf("Quantum degeneracy parameter: %f\n", in->Theta);
  printf("Quantum coupling parameter: %f\n", in->rs);
  printf("Chemical potential (low and high bound): %f %f\n", 
	 in->mu_lo, in->mu_hi);
  printf("Wave-vector cutoff: %f\n", in->xmax);
  printf("Wave-vector resolutions: %f\n", in->dx);
  printf("Number of Matsubara frequencies: %d\n", in->nl);
  printf("Maximum number of iterations: %d\n", in->nIter);
  printf("Error for convergence: %.5e\n", in->err_min_iter);
  printf("----------------------------------------------------\n");
 
  // Chemical potential
  if (verbose) printf("Chemical potential calculation: ");
  in->mu = compute_mu(*in);
  if (verbose) printf("Done. Chemical potential: %.8f\n", in->mu);
  
  // Wave-vector grid
  if (verbose) printf("Wave-vector grid initialization: ");
  wave_vector_grid(xx, *in);
  if (verbose) printf("Done.\n");
  
  // Normalized ideal Lindhard density
  if (verbose) printf("Normalized ideal Lindhard density calculation:\n");
  compute_phi(phi, xx, *in, verbose);
  if (verbose) printf("Done.\n");
  
  // Static structure factor in the Hartree-Fock approximation
  if (verbose) printf("Static structure factor in the Hartree-Fock approximation: ");
  compute_ssfHF(SSHF, xx, *in);
  if (verbose) printf("Done.\n");

}

// ------------------------------------------------------------------
// FUNCTION USED TO DEFINE THE WAVE-VECTOR GRID
// ------------------------------------------------------------------

void wave_vector_grid(float *xx, input in){
 
  xx[0] = in.dx/2.0;
  for (int ii=1; ii < in.nx; ii++) xx[ii] = xx[ii-1] + in.dx;

}

// -------------------------------------------------------------------
// FUNCTION USED TO ACCESS ONE ELEMENT OF A TWO-DIMENSIONAL ARRAY
// -------------------------------------------------------------------

int idx2(int xx, int yy, int x_size) {
  return (yy * x_size) + xx;
}


// -------------------------------------------------------------------
// FUNCTIONS USED TO COMPUTE THE NORMALIZED IDEAL LINDHARD DENSITY
// -------------------------------------------------------------------

void compute_phi(float *phi, float *xx,  input in, bool verbose) {

  // Threads and blocks
  int threadLimitPerBlock = 1024;
  int numberOfThreads = in.nx * in.nl;
  int numberOfBlocks = (numberOfThreads/threadLimitPerBlock) + 1;
  dim3 grid(numberOfBlocks, 1, 1);
  dim3 block(threadLimitPerBlock, 1, 1);

  // Arrays for device
  float *d_phi = NULL;
  float *d_xx = NULL;
  hipMalloc(&d_phi, sizeof(float) * in.nx * in.nl);
  hipMalloc(&d_xx, sizeof(float) * in.nx);

  // Copy arrays to device
  hipMemcpy(d_phi, psi, sizeof(float) * in.nx *in.nl, 
	     hipMemcpyHostToDevice);
  hipMemcpy(d_xx, xx, sizeof(float) * in.nx, 
	     hipMemcpyHostToDevice);  

  // Compute normalized ideal Lindhart density on device
  compute_phil <<< grid, block >>> (d_phi, d_xx, ll, in);
 
  // Copy solution from device 
  hipMemcpy(phi, d_phi, sizeof(float)*in.nx*in.nl, hipMemcpyDeviceToHost);

  // Free memory
  hipFree(d_phi);
  hipFree(d_xx);
 
}

__global__ void compute_phil(float *phil, float *xx,  int ll, input in) {

  int threadId = threadIdx.x + blockIdx.x * blockDim.x;

  if (threadId >= in.nx*in.nl) return;

  int ii = threadId % in.nx;
  int ll = threadId % in.nl;

  phil[threadId] = 0.0;
  if (ll == 0){
      
    for (int jj=0; jj<in.nx; jj++){
      phil[ii] += phix0(xx[jj], xx[ii], in);
    }
    phil[ii] *= in.dx;
    
  }
  else {
    
    for (int jj=0; jj<in.nx; jj++){
      phil[ii] += phixl(xx[jj], xx[ii], ll, in);
    }
    phil[ii] *= in.dx;
    

  }
    
  
}

__device__ float phixl(float yy, float xx, int ll, input in) {

  float yy2 = yy*yy, xx2 = xx*xx, txy = 2*xx*yy, 
    tplT = 2*M_PI*ll*in.Theta, tplT2 = tplT*tplT;

  if (xx > 0.0) {
    return 1.0/(2*xx)*yy/(exp(yy2/in.Theta - in.mu) + 1.0)
      *log(((xx2+txy)*(xx2+txy) + tplT2)/((xx2-txy)*(xx2-txy) + tplT2));
  }
  else {
    return 0;
  }

}

__device__ float phix0(float yy, float xx, input in) {

  float yy2 = yy*yy, xx2 = xx*xx, xy = xx*yy;

  if (xx > 0.0){

    if (xx < 2*yy){
      return 1.0/(in.Theta*xx)*((yy2 - xx2/4.0)*log((2*yy + xx)/(2*yy - xx)) + xy)
        *yy/(exp(yy2/in.Theta - in.mu) + exp(-yy2/in.Theta + in.mu) + 2.0);
    }
    else if (xx > 2*yy){
      return 1.0/(in.Theta*xx)*((yy2 - xx2/4.0)*log((2*yy + xx)/(xx - 2*yy)) + xy)
        *yy/(exp(yy2/in.Theta - in.mu) + exp(-yy2/in.Theta + in.mu) + 2.0);
    }
    else {
      return 1.0/(in.Theta)*yy2/(exp(yy2/in.Theta - in.mu)
                                 + exp(-yy2/in.Theta + in.mu) + 2.0);;
    }
  }

  else{
    return (2.0/in.Theta)*yy2/(exp(yy2/in.Theta - in.mu)
                               + exp(-yy2/in.Theta + in.mu) + 2.0);
  }

}

// -------------------------------------------------------------------
// FUNCTION USED TO COMPUTE THE STATIC STRUCTURE FACTOR
// -------------------------------------------------------------------

void compute_ssf(float *SS, float *SSHF, float *GG, 
		 float *phi, float *xx, input in){

  float lambda = pow(4.0/(9.0*M_PI), 1.0/3.0);
  float ff = 4*lambda*in.rs/M_PI;
  float xx2, BB, BB_tmp, BB_den, phixl;
  float tplT, Axl;

  for (int ii=0; ii<in.nx; ii++){

    if (xx[ii] > 0.0){
      xx2 = xx[ii]*xx[ii];
      BB = 0.0;
      
      for (int ll=0; ll<in.nl; ll++){
	tplT = 2*M_PI*ll*in.Theta;
	phixl = phi[idx2(ii,ll,in.nx)];
	Axl = (4.0/3.0)*xx2/(tplT*tplT + xx2*xx2);
	BB_den = 1.0 + ff/xx2*(1 - GG[ii])*phixl;
	//BB_tmp = phixl*phixl/BB_den - Axl*Axl;
	BB_tmp = phixl*phixl/BB_den;
	if (ll>0) BB_tmp *= 2.0;
	BB += BB_tmp;
	
      }
      
      /* SS[ii] = SSHF[ii] */
      /*   - 3.0/2.0*ff/xx2*in.Theta*(1- GG[ii])*BB */
      /*   - 1.0/3.0*ff/xx2/in.Theta*(1 - GG[ii])* */
      /*   (1.0/sinh(xx2/(2*in.Theta))* */
      /*    1.0/sinh(xx2/(2*in.Theta)) + */
      /*    2.0*in.Theta/xx2* */
      /*    1.0/tanh(xx2/(2*in.Theta))); */
      SS[ii] = SSHF[ii]
	- 3.0/2.0*ff/xx2*in.Theta*(1- GG[ii])*BB;
      
    }
    else 
      SS[ii] = 0.0;
  }

}

void compute_ssfHF(float *SS,  float *xx,  input in){

  // Static structure factor in the Hartree-Fock approximation
  for (int ii = 0; ii < in.nx; ii++) {

    SS[ii] = 0.0;
    for (int jj=0; jj<in.nx; jj++){
      SS[ii] += ssfHF(xx[jj], xx[ii], in);
    }
    SS[ii] *= in.dx;
    SS[ii] += 1.0;

  }
  
}
 
float ssfHF(float yy, float xx, input in) {

  float yy2 = yy*yy, ypx = yy + xx, ymx = yy - xx;
 
  if (xx > 0.0){
    return -3.0*in.Theta/(4.0*xx)*yy/(exp(yy2/in.Theta - in.mu) + 1.0)
      *log((1 + exp(in.mu - ymx*ymx/in.Theta))
           /(1 + exp(in.mu - ypx*ypx/in.Theta)));
  }
  else {
    return -3.0/2.0*yy2/(1.0 + cosh(yy2/in.Theta - in.mu));
  }


}

// -------------------------------------------------------------------
// FUNCTIONS USED TO COMPUTE THE STATIC LOCAL FIELD CORRECTION
// -------------------------------------------------------------------

void compute_slfc(float *GG, float *SS, float *xx, input in) {

  // Static local field correction
  for (int ii = 0; ii < in.nx; ii++) {
    
    GG[ii] = 0.0;
    for (int jj=0; jj<in.nx; jj++){
      GG[ii] += slfc(xx[jj], xx[ii], SS[jj]);
    }
    GG[ii] *= in.dx;

  }
  
}


float slfc(float yy, float xx, float SS) {

  float yy2 = yy * yy, xx2 = xx * xx;

  if (xx > 0.0 && yy > 0.0){
    
    if (xx > yy){
      return -(3.0/4.0) * yy2 * (SS - 1.0)
	* (1 + (xx2 - yy2)/(2*xx*yy)*log((xx + yy)/(xx - yy)));
    }
    else if (xx < yy) {
      return -(3.0/4.0) * yy2 * (SS - 1.0)
	* (1 + (xx2 - yy2)/(2*xx*yy)*log((xx + yy)/(yy - xx)));
    }
    else {
      return -(3.0/4.0) * yy2 * (SS - 1.0);
    }
    
  }
  else
    return 0;
  

}



// -------------------------------------------------------------------
// FUNCTIONS USED TO COMPUTE THE INTERNAL ENERGY
// -------------------------------------------------------------------

float compute_uex(float *SS, input in) {

  float ie;
  float lambda = pow(4.0/(9.0*M_PI), 1.0/3.0);  

  // Internal energy
  ie = 0.0;
  for (int jj=0; jj<in.nx; jj++){
    ie += SS[jj] - 1.0;
  }
  ie *= in.dx;

  // Output
  return ie/(M_PI*in.rs*lambda);

}

// -------------------------------------------------------------------
// FUNCTIONS FOR OUTPUT AND INPUT
// -------------------------------------------------------------------


// write text files for output
void write_text(float *SS, float *GG, float *phi, 
		float *SSHF, float *xx, input in){


    FILE* fid;
    
    // Output for SSF
    char out_name[100];
    sprintf(out_name, "ssf_rs%.3f_theta%.3f_%s.dat", in.rs, in.Theta, in.theory);
    fid = fopen(out_name, "w");
    if (fid == NULL) {
        perror("Error while creating the output file for the static structure factor");
        exit(EXIT_FAILURE);
    }
    for (int ii = 0; ii < in.nx; ii++)
        fprintf(fid, "%.8e %.8e\n", xx[ii], SS[ii]);

    fclose(fid);

    // Output for SLFC
    sprintf(out_name, "slfc_rs%.3f_theta%.3f_%s.dat", in.rs, in.Theta, in.theory);
    fid = fopen(out_name, "w");
    if (fid == NULL) {
        perror("Error while creating the output file for the static local field correction");
        exit(EXIT_FAILURE);
    }
    for (int ii = 0; ii < in.nx; ii++)
        fprintf(fid, "%.8e %.8e\n", xx[ii], GG[ii]);

    fclose(fid);

    // Output for static density response
    sprintf(out_name, "sdr_rs%.3f_theta%.3f_%s.dat", in.rs, in.Theta, in.theory);
    fid = fopen(out_name, "w");
    if (fid == NULL) {
      perror("Error while creating the output file for the static density response");
      exit(EXIT_FAILURE);
    }
    float lambda = pow(4.0/(9.0*M_PI), 1.0/3.0);
    float ff = 4*lambda*in.rs/M_PI;
    float sdr;
    for (int ii=0 ; ii<in.nx; ii++){
	sdr = -(3.0/2.0)*in.Theta*phi[idx2(ii,0,in.nx)]/
	  (1.0 + ff/(xx[ii]*xx[ii])*(1.0 - GG[ii])*phi[idx2(ii,0,in.nx)]);
	fprintf(fid, "%.8e %.8e\n", xx[ii], sdr);
      }
    fclose(fid);

    // Output for ideal Lindhard density response
    sprintf(out_name, "idr_rs%.3f_theta%.3f_%s.dat", in.rs, in.Theta, in.theory);
    fid = fopen(out_name, "w");
    if (fid == NULL) {
      perror("Error while creating the output file for the ideal density response");
      exit(EXIT_FAILURE);
    }
    for (int ii=0; ii<in.nx; ii++){
      for (int jj=0; jj<in.nl; jj++){
        fprintf(fid, "%.8e ", phi[idx2(ii,jj,in.nx)]);
      }
      fprintf(fid,"\n");
    }
    fclose(fid);

    // Output for static structure factor in the Hartree-Fock approximation
    sprintf(out_name, "ssfHF_rs%.3f_theta%.3f_%s.dat", in.rs, in.Theta, in.theory);
    fid = fopen(out_name, "w");
    if (fid == NULL) {
        perror("Error while creating the output file for the static structure factor (HF)");
        exit(EXIT_FAILURE);
    }
    for (int ii = 0; ii < in.nx; ii++)
        fprintf(fid, "%.8e %.8e\n", xx[ii], SSHF[ii]);

    fclose(fid);

}


// write binary file to use as initial guess (or restart)
void write_guess(float *SS, float *GG, input in){

  // Name of output file
  char out_name[100];
  sprintf(out_name, "restart_rs%.3f_theta%.3f_%s.bin", in.rs, in.Theta, in.theory);

  // Open binary file
  FILE *fid = NULL;
  fid = fopen(out_name, "wb");
  if (fid == NULL) {
    fprintf(stderr,"Error while creating file for restart");
    exit(EXIT_FAILURE);
  }

  // Static structure factor 
  fwrite(&in, sizeof(input), 1, fid);

  // Static structure factor 
  fwrite(SS, sizeof(float), in.nx, fid);

  // Static local field correction
  fwrite(GG, sizeof(float), in.nx, fid);

  // Close binary file
  fclose(fid);

}


// read binary file to use as initial guess (or restart)
void read_guess(float *SS, float *GG, input in){

  // Variables
  input in_load;

  // Open binary file
  FILE *fid = NULL;
  fid = fopen(in.guess_file, "rb");
  if (fid == NULL) {
    fprintf(stderr,"Error while opening file with density response");
    exit(EXIT_FAILURE);
  }

  // Check that the data for the guess file is consistent
  fread(&in_load, sizeof(input), 1, fid);
  if (in_load.nx != in.nx || in_load.dx != in.dx || in_load.xmax != in.xmax){
    fprintf(stderr,"Grid from guess file is incompatible with input\n");
    fclose(fid);
    exit(EXIT_FAILURE);
  }
  
  // Static structure factor in the Hartree-Fock approximation
  fread(SS, sizeof(float), in_load.nx, fid);

  // Static structure factor in the Hartree-Fock approximation
  fread(GG, sizeof(float), in_load.nx, fid);

  // Close binary file
  fclose(fid);
	    
}
