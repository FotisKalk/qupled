#include "hip/hip_runtime.h"
extern "C" {
#include <stdio.h>
#include <stdbool.h>
#include <time.h>
#include <getopt.h>
#include <string.h>
#include <errno.h>
#include <gsl/gsl_math.h>
#include <gsl/gsl_errno.h> 
#include <gsl/gsl_roots.h>
#include <gsl/gsl_sf_gamma.h>
#include <gsl/gsl_sf_fermi_dirac.h>
#include "stls.h"
#include "qstls.h"
#include "qstls_gpu.h"
}

// -------------------------------------------------------------------
// FUNCTION USED TO ITERATIVELY SOLVE THE STLS-HNC EQUATIONS
// -------------------------------------------------------------------

void solve_qstls(input in, bool verbose) {

  // Arrays for STLS solution
  float *xx = NULL; 
  float *phi = NULL;
  float *psi = NULL;
  float *SS = NULL;
  float *SS_new = NULL;
  float *SSHF = NULL;
  float *GG = NULL;
  float *GG_new = NULL;

  // Solve STLS equation for initial guess
  if (verbose) printf("Solution of classical STLS for initial guess:\n");
  float a_mix_hold = in.a_mix;
  in.a_mix = 0.1;
  solve_stls(in, false, &xx, &SS, &SSHF, &GG, &GG_new, &phi);
  free(GG);
  free(GG_new);
  in.a_mix = a_mix_hold;
  if (verbose) printf("Done.\n");

  // Allocate arrays for qstls calculation
  alloc_qstls_arrays(in, &psi, &SS_new);  
  
  // SSF and SLFC via iterative procedure
  if (verbose) printf("SSF and SLFC calculation...\n");
  float iter_err = 1.0;
  int iter_counter = 0;
  while (iter_counter < in.nIter && iter_err > in.err_min_iter ) {
    
    // Start timing
    clock_t tic = clock();
    
    // Update auxilliary response 
    compute_psi(psi, xx, SS, in);
    
    // Update SSF
    compute_qstls_ssf(SS_new, SSHF, phi, psi, xx, in);

    // Update diagnostic
    iter_err = 0.0;
    iter_counter++;
    for (int ii=0; ii<in.nx; ii++) {
      iter_err += (SS_new[ii] - SS[ii]) * (SS_new[ii] - SS[ii]);
      SS[ii] = in.a_mix*SS_new[ii] + (1-in.a_mix)*SS[ii];
    }
    iter_err = sqrt(iter_err);
        
    // End timing
    clock_t toc = clock();
    
    // Print diagnostic
    if (verbose) {
      printf("--- iteration %d ---\n", iter_counter);
      printf("Elapsed time: %f seconds\n", ((float)toc - (float)tic) / CLOCKS_PER_SEC);
      printf("Residual error: %.5e\n", iter_err);
      fflush(stdout);
    }
  }
  if (verbose) printf("Done.\n");
  
  // Internal energy
  if (verbose) printf("Internal energy: %f\n",compute_uex(SS, in));
  
  // Output to file
  if (verbose) printf("Writing output files...\n");
  write_text_qstls(SS, psi, xx, in);
  if (verbose) printf("Done.\n");

  // Output to variable or free memory
  free_qstls_arrays(xx, phi, psi, SS, SS_new, SSHF);
 
 
}


// -------------------------------------------------------------------
// FUNCTIONS USED TO ALLOCATE AND FREE ARRAYS
// -------------------------------------------------------------------

void alloc_qstls_arrays(input in, float **psi, float **SS_new){

  *psi = (float*)malloc( sizeof(float) * in.nx * in.nl);
  *SS_new = (float*)malloc( sizeof(float) * in.nx);

}

void free_qstls_arrays(float *xx, float *phi,
                      float *psi, float *SS,
                      float *SS_new, float *SSHF){

  free(xx);
  free(phi);
  free(psi);
  free(SSHF);
  free(SS);
  free(SS_new);

}


// -------------------------------------------------------------------
// FUNCTIONS USED TO COMPUTE THE AUXILLIARY RESPONSE
// -------------------------------------------------------------------

void compute_psi(float *psi, float *xx,  float *SS, input in) {

  // Threads and blocks
  int threadLimitPerBlock = 1024;
  int numberOfThreads = in.nx * in.nl;
  int numberOfBlocks = (numberOfThreads/threadLimitPerBlock) + 1;
  dim3 grid(numberOfBlocks, 1, 1);
  dim3 block(threadLimitPerBlock, 1, 1);

  // Arrays for device
  float *d_psi = NULL;
  float *d_xx = NULL;
  float *d_SS = NULL;
  hipMalloc(&d_psi, sizeof(float) * in.nx * in.nl);
  hipMalloc(&d_xx, sizeof(float) * in.nx);
  hipMalloc(&d_SS, sizeof(float) * in.nx);

  // Copy arrays to device
  hipMemcpy(d_psi, psi, sizeof(float) * in.nx *in.nl, 
	     hipMemcpyHostToDevice);
  hipMemcpy(d_xx, xx, sizeof(float) * in.nx, 
	     hipMemcpyHostToDevice);  
  hipMemcpy(d_SS, SS, sizeof(float) * in.nx, 
	     hipMemcpyHostToDevice);

  // Launch kernel on device
  compute_psil <<< grid, block >>> (d_psi, d_xx, d_SS, in);
 
  // Copy solution from device 
  hipMemcpy(psi, d_psi, sizeof(float)*in.nx*in.nl, hipMemcpyDeviceToHost);

  // Free memory
  hipFree(d_psi);
  hipFree(d_xx);
  hipFree(d_SS);

}

__global__ void compute_psil(float *psi, float *xx, float *SS, input in) {

  int threadId = threadIdx.x + blockIdx.x * blockDim.x;

  if (threadId >= in.nx*in.nl) return;

  float nu = (int)floor(2.0/in.dx);
  float fw, fq;
  int ii = threadId % in.nx;
  int ll = threadId / in.nl;

  psi[threadId] = 0.0;
  for (int jj=0; jj<in.nx; jj++){
      
    fw = 0.0;

    for (int kk=0; kk<in.nx; kk++){

      fq = 0.0;
	
      for (int mm=0; mm<nu; mm++){
  	fq += psi_u(-1.0 + mm*in.dx, xx[kk], xx[jj],
  		    xx[ii], ll, in);
      }

      fw += fq*in.dx*psi_q(xx[kk], ll, in);
		  
    }

    psi[threadId] += fw*in.dx*psi_w(xx[ii], SS[ii]);
    
  }
  
  if (ll == 0) psi[threadId] *= -(3.0/8.0)*in.dx;
  else psi[threadId] *= -3.0/(4.0*in.Theta)*in.dx;
  
}



__device__ float psi_u(float uu, float qq, float ww,
	     float xx, int ll, input in){

  float xx2 = xx*xx, qq2 = qq*qq, ww2 = ww*ww, 
    txq = 2.0*xx*qq, tplT = 2*M_PI*ll*in.Theta, 
    tplT2 = tplT*tplT, xwu = xx*ww*uu, 
    tt = xx2 - xwu, tt2 = tt*tt, 
    fact = ww*xx/(ww2 + tt - xwu);
  float logarg;
    
  if (ll == 0){

    if (tt == txq || tt == -txq) {
      return fact * 2.0 * qq*tt/xx;
    }
    else {
      
      logarg = (tt + txq)/(tt - txq);
      if (logarg < 0) logarg = -logarg;
      return fact * ((qq2 - tt2/(4.0*xx2))* 
      		     log(logarg) + qq*tt/xx);
    }

  }

  else {

    return fact * log(((txq + tt)*(txq + tt) + tplT2)
		      /((txq - tt)*(txq - tt) + tplT2));

  }

}

__device__ float psi_q(float qq, int ll, input in){

  float qq2 = qq*qq;

  if (ll == 0){

    return qq/(exp(qq2/in.Theta - in.mu) 
		+ exp(-qq2/in.Theta + in.mu) + 2.0);

  }
  else {

    return qq/(exp(qq2/in.Theta - in.mu) +  1.0);

  }

}

__device__ float psi_w(float ww, float SS){

  return ww * (SS - 1);
  
}

// -------------------------------------------------------------------
// FUNCTION USED TO COMPUTE THE STATIC STRUCTURE FACTOR
// -------------------------------------------------------------------

void compute_qstls_ssf(float *SS, float *SSHF, float *phi,
		       float *psi, float *xx, input in){

  float lambda = pow(4.0/(9.0*M_PI), 1.0/3.0);
  float pilambda = M_PI*lambda;
  float ff = 4*lambda*lambda*in.rs;
  float ff3_2T = 3.0*in.Theta*ff/2.0;
  float xx2, BB, BB_tmp, phixl, psixl;
  for (int ii=0; ii<in.nx; ii++){

    xx2 = xx[ii]*xx[ii];
    BB = 0.0;

    for (int ll=0; ll<in.nl; ll++){
      phixl = phi[idx2(ii,ll,in.nx)];
      psixl = psi[idx2(ii,ll,in.nx)];
      BB_tmp = phixl*(phixl - psixl)/
	(pilambda*xx2 + ff*(phixl-psixl));
      if (ll>0) BB_tmp *= 2.0;
      BB += BB_tmp;
    }
    
    SS[ii] = SSHF[ii] - ff3_2T*BB;

  }

}

// -------------------------------------------------------------------
// FUNCTIONS FOR OUTPUT AND INPUT
// -------------------------------------------------------------------


// write text files with SSF and SLFC
void write_text_qstls(float *SS, float *psi, float *xx, input in){


    FILE* fid;
    
    // Output for SSF
    fid = fopen("ssf_QSTLS.dat", "w");
    if (fid == NULL) {
        perror("Error while creating the output file for the static structure factor");
        exit(EXIT_FAILURE);
    }
    for (int ii=0; ii<in.nx; ii++)
    {
        fprintf(fid, "%.8e %.8e\n", xx[ii], SS[ii]);
    }
    fclose(fid);

    // Output for auxilliary response
    fid = fopen("psi_QSTLS.dat", "w");
    if (fid == NULL) {
        perror("Error while creating the output file for the auxilliary response");
        exit(EXIT_FAILURE);
    }
    for (int ii=0; ii<in.nx; ii++){
      for (int jj=0; jj<in.nl; jj++){
        fprintf(fid, "%.8e ", psi[idx2(ii,jj,in.nx)]);
      }
      fprintf(fid,"\n");
    }
    fclose(fid);


}

